#include<stdio.h>
#include<hip/hip_runtime.h>

void myCudaCheck(hipError_t err) {
        if(err != hipSuccess) {
                printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);
                exit(EXIT_FAILURE);
        }
}

void fill_matrix(double *mat, unsigned numRows, unsigned numCols)
{
	for(unsigned i=0; i < numRows; i++)
	for(unsigned j=0; j < numCols; j++)
	{
		mat[i*numCols + j] = i*2.1f + j*3.2f;
	}	
}


void print_matrix_to_file(double *mat, unsigned numRows, unsigned numCols)
{
	const char *fname = "assignment2_out";
	FILE *f = fopen(fname, "a");
	for(unsigned i=0; i < numRows; i++)	
	{
		for(unsigned j=0; j < numCols; j++)
			fprintf(f,"%4.4f ", mat[i*numCols + j]);
		fprintf(f,"\n");
	}
	fclose(f);
}


__global__ void matrixMultiply1(double *A, double *B, double *C, unsigned N ){

	int row, col;
	row = blockIdx.x*blockDim.x + threadIdx.x;
	col = blockIdx.y*blockDim.y + threadIdx.y;
	
	C[ row*N + col ] = 0; 
	for( int i = 0; i < N; i++ ){
		C[ row*N + col ] += A[ row*N + i ]*B[ i*N + col ];
	}

}
__global__ void matrixMultiply2(double *A, double *B, double *C, unsigned N ){

	int row, col;
	col = blockIdx.x*blockDim.x + threadIdx.x;
	row = blockIdx.y*blockDim.y + threadIdx.y;
	
	C[ row*N + col ] = 0;
	for( int i = 0; i < N; i++ ){
		C[ row*N + col ] += A[ row*N + i ]*B[ i*N + col ];
	}

}

int main(){

	unsigned N = 8192;  //size of matrix
	size_t mat_size = sizeof(double)*N*N;
	
	double *d_A, *d_B, *d_C;

	double *h_A = (double *) malloc( mat_size );
	double *h_B = (double *) malloc( mat_size );
	double *h_C = (double *) malloc( mat_size );
	
	dim3 threads( 16, 16 );
	dim3 blocks( N/16, N/16 );

	hipEvent_t start1, stop1, start2, stop2;
	
	hipEventCreate( &start1 );
	hipEventCreate( &stop1 );
	hipEventCreate( &start2 );
	hipEventCreate( &stop2 );
	
	fill_matrix( h_A, N, N );
	fill_matrix( h_B, N, N );
	fill_matrix( h_C, N, N );

	myCudaCheck( hipMalloc( &d_A, mat_size ) );
	myCudaCheck( hipMalloc( &d_B, mat_size ) );
	myCudaCheck( hipMalloc( &d_C, mat_size ) );

	myCudaCheck( hipMemcpy( d_A, h_A, mat_size, hipMemcpyHostToDevice ) );
	myCudaCheck( hipMemcpy( d_B, h_B, mat_size, hipMemcpyHostToDevice ) );

	hipEventRecord( start1 );
	matrixMultiply1<<< blocks, threads >>>( d_A, d_B, d_C, N );
	hipEventRecord( stop1 );
	
	myCudaCheck( hipMemcpy( h_C, d_C, mat_size, hipMemcpyDeviceToHost ) );
	print_matrix_to_file( h_C, N, N );
	
	hipEventRecord( start2 );
	matrixMultiply2<<< blocks, threads >>>( d_A, d_B, d_C, N );
	hipEventRecord( stop2 );

	myCudaCheck( hipMemcpy( h_C, d_C, mat_size, hipMemcpyDeviceToHost ) );
	
	hipEventSynchronize( stop1 );
	hipEventSynchronize( stop2 );
	
	print_matrix_to_file( h_C, N, N );
	
	float t1,t2;
	hipEventElapsedTime( &t1, start1, stop1 );
	hipEventElapsedTime( &t2, start2, stop2 );
	
	printf("%f\n", t1);
	printf("%f\n", t2);
	
	return 0;
}
